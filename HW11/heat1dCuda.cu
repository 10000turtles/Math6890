
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <float.h>
#include <assert.h>
#include <ctime>
#include <string>
#include <iostream>

using std::max;
using std::string;
using namespace std;

typedef double Real;

#define x(i) x_p[i - nd1a]
#define boundaryCondition(side, axis) boundaryCondition_p[(side) + 2 * (axis)]
#define UTRUE(x, t) (b0 + (x) * (b1 + (x) * b2)) * (a0 + (t) * (a1))
#define UTRUEX(x, t) (b1 + 2. * (x) * b2) * (a0 + (t) * (a1))
#define UTRUET(x, t) (b0 + (x) * (b1 + (x) * b2)) * (a1)
#define UTRUEXX(x, t) (2. * b2) * (a0 + (t) * (a1))
#define FORCE(x, t) (UTRUET(x, t) - kappa * UTRUEXX(x, t))

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s atline %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void time_step(Real *u_c, Real *u_n, Real *x_p, Real *params)
{
#define uc(i) u_c[i - nd1a]
#define un(i) u_n[i - nd1a]

    Real b0 = 1., b1 = .5, b2 = .25;
    Real a0 = 1., a1 = .3, kappa = .1;

    Real rx = params[0];
    Real t = params[1];
    Real dt = params[2];
    int n1a = params[3];
    int n1b = params[4];
    int nd1a = params[5];
    int nd1b = params[6];

    int i = n1a + threadIdx.x + blockIdx.x * blockDim.x;
    int n = n1b - n1a + 1;

    if (i < n)
    {
        un(i) = uc(i) + rx * (uc(i + 1) - 2. * uc(i) + uc(i - 1)) + dt * FORCE(x(i), t);
    }
}

__global__ void boundary_conditions(Real *u_c, Real *u_n, Real *x_p, Real *params)
{
    Real b0 = 1., b1 = .5, b2 = .25;
    Real a0 = 1., a1 = .3, kappa = .1;

    Real rx = params[0];
    Real t = params[1];
    Real dt = params[2];
    int n1a = params[3];
    int n1b = params[4];
    int nd1a = params[5];
    int nd1b = params[6];

    for (int side = 0; side <= 1; side++)
    {
        const int i = side == 0 ? n1a : n1b; // boundaryindex
        const int is = 1 - 2 * side;         // is=1onleft,-1onright

        un(i) = UTRUE(x(i), t + dt);
        un(i - is) = UTRUE(x(i - is), t + dt); // extrapolateghost
    }
    params[1] = t + dt;

    return;
}

__global__ void update_current(Real *u_c, Real *u_n, Real *params)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int nd1 = params[7];
    if (i < nd1)
    {
        u_c[i] = u_n[i];
    }
}

#define uc(i) u_p[cur][i - nd1a]
#define un(i) u_p[next][i - nd1a]

int main(int argc, char *argv[])
{
    // Setup

    const Real pi = M_PI;

    int debug = 0; // setto1fordebuginfo
    Real xa = 0., xb = 1.;
    Real kappa = .1;
    Real tFinal = atof(argv[2]);
    Real cfl = .9; // time-stepsafetyfactor

    int Nx = 10; // default
    string matlabFileName = "heat1d.m";

    if (argc >= 2) // readanycommandlinearguments
    {
        Nx = atoi(argv[1]);
        printf("SettingNx=%d\n", Nx);
    }

    Real dx = (xb - xa) / Nx;
    const int numGhost = 1;
    const int n1a = 0;
    const int n1b = Nx;
    const int nd1a = n1a - numGhost;
    const int nd1b = n1b + numGhost;
    const int nd1 = nd1b - nd1a + 1; // totalnumberofgridpoints;

    Real *x_p = new Real[nd1];

    for (int i = nd1a; i <= nd1b; i++)
        x(i) = xa + (i - n1a) * dx;

    if (debug > 1)
    {
        for (int i = nd1a; i <= nd1b; i++)
            printf("x(%2d)=%12.4e\n", i, x(i));
    }

    const int dirichlet = 1, neumann = 2;
    const int numberOfDimensions = 1;
    int *boundaryCondition_p = new int[2 * numberOfDimensions];

    const Real kx = 3.;
    const Real kxPi = kx * pi;
    const Real kappaPiSq = kappa * kxPi * kxPi;

    const char solutionName[] = "polyDD";
    boundaryCondition(0, 0) = dirichlet;
    boundaryCondition(1, 0) = dirichlet;

    Real b0 = 1., b1 = .5, b2 = .25;
    Real a0 = 1., a1 = .3;

    Real *u_p[2];
    u_p[0] = new Real[nd1];
    u_p[1] = new Real[nd1];

    // initialconditions
    Real t = 0.;
    int cur = 0; //"current"solution,indexintou_p[]
    for (int i = nd1a; i <= nd1b; i++)
        uc(i) = UTRUE(x(i), t);

    const Real dx2 = dx * dx;
    Real dt = cfl * .5 * dx2 / kappa; // dt,adjustedbelow
    const int numSteps = ceil(tFinal / dt);
    dt = tFinal / numSteps; // adjustdttoreachthefinaltime
    const Real rx = kappa * dt / dx2;

    // Declare GPU vars
    Real *gpu_u_p_c;
    Real *gpu_u_p_n;
    Real *gpu_x_p;

    Real params[8];
    params[0] = rx;
    params[1] = t;
    params[2] = dt;
    params[3] = n1a;
    params[4] = n1b;
    params[5] = nd1a;
    params[6] = nd1b;
    params[7] = nd1;

    Real *gpu_params;

    HANDLE_ERROR(hipMalloc((void **)&gpu_u_p_c, nd1 * sizeof(Real)));
    HANDLE_ERROR(hipMalloc((void **)&gpu_u_p_n, nd1 * sizeof(Real)));
    HANDLE_ERROR(hipMalloc((void **)&gpu_x_p, nd1 * sizeof(Real)));
    HANDLE_ERROR(hipMalloc((void **)&gpu_params, 8 * sizeof(Real)));

    HANDLE_ERROR(hipMemcpy(gpu_u_p_c, u_p[0], nd1 * sizeof(Real), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(gpu_u_p_n, u_p[1], nd1 * sizeof(Real), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(gpu_x_p, x_p, nd1 * sizeof(Real), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(gpu_params, params, 8 * sizeof(Real), hipMemcpyHostToDevice));

    hipEvent_t start;
    hipEvent_t end;
    float time = 0;

    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);

    // Run Time Stepping Loop
    for (int n = 0; n < numSteps; n++)
    {
        t = n * dt;

        const int cur = n % 2;
        const int next = (n + 1) % 2;

        int Nt = 128;
        long int Nb = ceil((float)Nx / Nt);

        time_step<<<Nb, Nt>>>(gpu_u_p_c, gpu_u_p_n, gpu_x_p, gpu_params);
        hipDeviceSynchronize();
        boundary_conditions<<<1, 1>>>(gpu_u_p_c, gpu_u_p_n, gpu_x_p, gpu_params);
        hipDeviceSynchronize();
        Nt = 128;
        Nb = ceil((float)nd1 / Nt);
        update_current<<<Nb, Nt>>>(gpu_u_p_c, gpu_u_p_n, gpu_params);
        hipDeviceSynchronize();
    }
    t += dt;

    hipEventRecord(end, 0);

    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, end);

    // Bring back data

    Real *u_p_new = new Real[nd1];

    HANDLE_ERROR(hipMemcpy(u_p_new, gpu_u_p_n, nd1 * sizeof(Real), hipMemcpyDeviceToHost));

    // Ending stuff

    Real cpuTimeStep = time / 1000;
    Real *error_p = new Real[nd1];
#define error(i) error_p[i - nd1a]

    cur = 0;
    Real maxErr = 0.;
    for (int i = nd1a; i <= nd1b; i++)
    {
        error(i) = u_p_new[i + 1] - UTRUE(x(i), t);
        maxErr = max(maxErr, abs(error(i)));
    }

    printf("numSteps=%4d,Nx=%3d,maxErr=%9.2e,gpu=%9.2e(s)\n", numSteps, Nx, maxErr, cpuTimeStep);
}