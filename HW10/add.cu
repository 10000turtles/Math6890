
// export PATH=/usr/local/cuda/bin:$PATH


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <ctime>

using namespace std;

static void HandleError(hipError_t err, const char *file, int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s atline %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

__global__ void add_vector(long int *a, long int *b, long int *c, long int *n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    i = i % *n;
    if (i < *n)
        c[i] = a[i] + b[i];
}

int main(int args, char **argv)
{
    long int n = atoi(argv[1]);
    long int Nt = 1;
    float first_time;

    for (int count = 0; count <= 10; count++)
    {
        long int Nb = ceil((float)n / Nt);

        long int *a = new long int[n];
        long int *b = new long int[n];
        long int *c = new long int[n];

        for (long int i = 0; i < n; i++)
        {
            a[i] = -i;
            b[i] = i * i;
            c[i] = 0;
        }

        // Cuda code
        long int *gpu_a;
        long int *gpu_b;
        long int *gpu_c;
        long int *gpu_n;

        HANDLE_ERROR(hipMalloc((void **)&gpu_a, n * sizeof(long int)));

        HANDLE_ERROR(hipMalloc((void **)&gpu_b, n * sizeof(long int)));
        HANDLE_ERROR(hipMalloc((void **)&gpu_c, n * sizeof(long int)));
        HANDLE_ERROR(hipMalloc((void **)&gpu_n, sizeof(long int)));

        HANDLE_ERROR(hipMemcpy(gpu_a, a, n * sizeof(long int), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(gpu_b, b, n * sizeof(long int), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(gpu_c, c, n * sizeof(long int), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(gpu_n, &n, sizeof(long int), hipMemcpyHostToDevice));

        hipEvent_t start;
        hipEvent_t end;
        float time = 0;

        hipEventCreate(&start);
        hipEventCreate(&end);

        hipEventRecord(start, 0);
        add_vector<<<Nb, Nt>>>(gpu_a, gpu_b, gpu_c, gpu_n);
        hipEventRecord(end, 0);

        hipDeviceSynchronize();
        hipEventElapsedTime(&time, start, end);

        HANDLE_ERROR(hipMemcpy(c, gpu_c, n * sizeof(long int), hipMemcpyDeviceToHost));

        long int error = 0;

        if (count == 0)
            first_time = (1.0 * std::clock()) / CLOCKS_PER_SEC;

        for (long int i = 0; i < n; i++)
        {
            c[i] = c[i] - a[i] - b[i];
            error = error + c[i];
        }

        if (count == 0)
            first_time = (1.0 * std::clock()) / CLOCKS_PER_SEC - first_time;

        cout << Nb << " " << Nt << " " << Nb * Nt << " " << n << " " << error << " " << time << " " << first_time / time * 1000 << endl;

        free(a);
        free(b);
        free(c);

        hipFree(gpu_a);
        hipFree(gpu_b);
        hipFree(gpu_c);

        Nt = Nt * 2;
    }
    return 0;
}
